#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hip/hip_fp16.h>

__global__
void force_solve_cheap_sqrt(float* pos, float* mass, float* acc_phi, float G, float eps, int n_particles){
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = 0;

    float pos_ix = pos[i*3];
    float pos_iy = pos[i*3 + 1];
    float pos_iz = pos[i*3 + 2];

    float pos_jx;
    float pos_jy;
    float pos_jz;

    float diffx;
    float diffy;
    float diffz;

    __half hdiffx;
    __half hdiffy;
    __half hdiffz;

    float ax = 0;
    float ay = 0;
    float az = 0;
    float gpe = 0;

    float mass_i = mass[i];
    float mass_j;

    float dist;

    float acc_mul;

    __half htemp0;
    __half htemp1;
    __half htemp2;
    __half htemp3;

    float temp0;
    float temp1;
    
    for (j = 0; j < n_particles; j++){

        if (j != i) {
            pos_jx = pos[j*3];
            pos_jy = pos[j*3 + 1];
            pos_jz = pos[j*3 + 2];

            mass_j = mass[j];

            diffx = pos_jx - pos_ix;
            diffy = pos_jy - pos_iy;
            diffz = pos_jz - pos_iz;

            hdiffx = __float2half(diffx);
            hdiffy = __float2half(diffy);
            hdiffz = __float2half(diffz);

            htemp0 = __hmul(hdiffx,hdiffx);
            htemp1 = __hmul(hdiffy,hdiffy);
            htemp2 = __hmul(hdiffz,hdiffz);

            htemp3 = __hadd(htemp0,htemp1);
            htemp0 = __hadd(htemp3,htemp2);

            htemp2 = hsqrt(htemp0);
            dist = __half2float(htemp2);

            temp0 = dist*dist;
            temp1 = temp0 * dist;
            temp0 = G * mass_j;

            acc_mul = temp0/(temp1);

            temp0 = acc_mul * diffx;
            ax = ax + temp0;

            temp0 = acc_mul * diffy;
            ay = ay + temp0;

            temp0 = acc_mul * diffz;
            az = az + temp0;

            temp0 = mass_i * mass_j;
            temp1 = temp0 * G;
            temp0 = temp1 * -1;

            gpe = gpe + (temp0 / dist);
        }

    }

    acc_phi[i*4] = ax;
    acc_phi[i*4 + 1] = ay;
    acc_phi[i*4 + 2] = az;
    acc_phi[i*4 + 3] = gpe;
    
}