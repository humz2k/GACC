#include "hip/hip_runtime.h"
#include "kernels.h"

__global__
void force_solve_cheap_shared_mem(half2* pos, half* mass, float* acc_phi, float G, float eps, int n_particles){

     //TODO: use local memory: make shared memory array of size blocksize. 

    
    extern __shared__ half2 s[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j;
    int k;

    __half2 h2G = __float2half2_rn(G);
    __half heps = __float2half(eps);

    __half2 h2pos_ixy = pos[i*2];
    __half hpos_iz = __high2half(pos[i*2 + 1]);

    __half hmass_i = mass[i];
    __half2 h2mass_j;

    __half2 h2temp0;
    __half2 h2temp1;
    __half2 h2temp2;

    __half htemp0 = 1;
    __half htemp1 = 2;
    __half htemp2 = 3;
    __half htemp3 = 4;

    __half2 h2diffxy;
    __half hdiffz;

    __half2 h2axy = __float2half2_rn(0);
    __half2 h2az_gpe = __float2half2_rn(0);

    __half2 h2dist;
    __half2 h2Gmul;
    __half2 h2acc_mul;

    int n_repeats = n_particles / blockDim.x;
    //int laneID = threadIdx.x % 32;

    for (j = 0; j < n_repeats; j++){

        int startIdx = j * blockDim.x;
        int myIdx = threadIdx.x + startIdx;
        s[threadIdx.x * 2] = pos[myIdx * 2];
        s[threadIdx.x * 2 + 1] = __halves2half2(__high2half(pos[myIdx * 2 + 1]),mass[myIdx]);

        __syncthreads();
        
        for (k = 0; k < blockDim.x; k++){

            if (k + startIdx != i){

                if (__hne(hmass_i,0)){

                    h2temp0 = s[k*2];
                    htemp0 = __low2half(s[k*2 + 1]);

                    h2mass_j = __high2half2(s[k*2 + 1]);

                    h2diffxy = __hsub2(h2temp0,h2pos_ixy); //gets difference of jxy and ixy into half2
                    hdiffz = __hsub(htemp0,hpos_iz); //gets difference of jz and iz into half

                    h2temp0 = __hmul2(h2diffxy,h2diffxy); //squares diffxy
                    htemp0 = __hfma(hdiffz,hdiffz,heps); //squars diffz and adds smoothing eps

                    htemp1 = __low2half(h2temp0); //divides xy into two halfs
                    htemp2 = __high2half(h2temp0);

                    htemp3 = __hadd(htemp0,htemp1); //adds (diffz**2 + eps) and diffx**2
                    htemp0 = __hadd(htemp3,htemp2); //adds (diffz**2 + eps + diffx**2) and diffy**2

                    h2temp1 = __half2half2(htemp0); //copies half (diffz**2 + eps + diffx**2 + diffy**2) to a half2 (distance**2)

                    h2dist = h2sqrt(h2temp1); //square roots half (diffz**2 + eps + diffx**2 + diffy**2) to a half2

                    h2temp0 = __hmul2(h2temp1,h2dist); //multiplies half2 distance by half2 distance**2 to get half2 distance**3

                    h2Gmul = __hmul2(h2G,h2mass_j); //multiplies G by mass_j to half2
                    h2acc_mul = __h2div(h2Gmul,h2temp0); //divides by distance**3

                    htemp0 = __high2half(h2temp1); //calculates mass_i * distance**3 and stores in a half
                    htemp1 = __hmul(htemp0,hmass_i);

                    h2temp2 = __halves2half2(hdiffz,htemp1); //combines diffz and massi*distance**3 into one half2

                    h2temp0 = __hmul2(h2acc_mul,h2diffxy); //calculates acceleration for xy
                    h2temp1 = __hmul2(h2acc_mul,h2temp2); //calculates acceleration for z and multiplies (G * mj / d**3) by mass_i * d**2 for gpe

                    h2axy = __hadd2(h2axy,h2temp0); //adds acceleration of xy to h2axy
                    h2az_gpe = __hadd2(h2az_gpe,h2temp1); //adds acceleation of z and gpe to h2az_gpe
                }

            }

        }

    }

    acc_phi[i*4] = __low2float(h2axy);
    acc_phi[i*4 + 1] = __high2float(h2axy);
    acc_phi[i*4 + 2] = __low2float(h2az_gpe);
    acc_phi[i*4 + 3] = (-1) * __high2float(h2az_gpe);

}