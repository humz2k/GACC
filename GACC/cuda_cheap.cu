#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <omp.h>
#include "cuda_cheap.h"
#include "cheap_kernels/kernels.h"
#include <hip/hip_fp16.h>

using namespace std;

void save(float* pos, float* vel, float* phi_acc, int n_particles, std::ofstream &out){

    for (int i = 0; i < n_particles; i++){

        out.write( reinterpret_cast<char*>( &pos[i*3]), sizeof( float ));
        out.write( reinterpret_cast<char*>( &pos[i*3 + 1]), sizeof( float ));
        out.write( reinterpret_cast<char*>( &pos[i*3 + 2]), sizeof( float ));

        out.write( reinterpret_cast<char*>( &vel[i*3]), sizeof( float ));
        out.write( reinterpret_cast<char*>( &vel[i*3 + 1]), sizeof( float ));
        out.write( reinterpret_cast<char*>( &vel[i*3 + 2]), sizeof( float ));

        out.write( reinterpret_cast<char*>( &phi_acc[i*4]), sizeof( float ));
        out.write( reinterpret_cast<char*>( &phi_acc[i*4 + 1]), sizeof( float ));
        out.write( reinterpret_cast<char*>( &phi_acc[i*4 + 2]), sizeof( float ));
        out.write( reinterpret_cast<char*>( &phi_acc[i*4 + 3]), sizeof( float ));

    }

}

extern "C" { 
    void cuda_evaluate(float* input_pos, float* input_vel, float* input_mass, int n_particles, int steps, float G, float eps, float dt, int n_params, int solver, int v, double *saveTime, double *totalTime, double *copyTime){

        double first,second;
        double total_first,total_second;

        hipFree(0);
        hipDeviceSynchronize();

        total_first = omp_get_wtime();

        *saveTime = 0;
        *totalTime = 0;
        *copyTime = 0;

        std::ofstream out;
        out.open( "out.dat", std::ios::out | std::ios::binary);
        std::ofstream &fp = out;

        int blockSize = n_particles;
        if (blockSize > 256){
            
            if ((n_particles % 256) == 0){
                blockSize = 256;
            } else if ((n_particles % 128) == 0){
                blockSize = 128;
            } else if ((n_particles % 64) == 0){
                blockSize = 64;
            } else if ((n_particles % 32) == 0){
                blockSize = 32;
            }

        }
        int numBlocks = (n_particles + blockSize - 1) / blockSize;

        if (v){
            cout << "numBlocks" << numBlocks << endl;
            cout << "blockSize" << blockSize << endl;
        }
        
        half *d_hmass;
        half2 *d_h2pos;
        float *d_pos, *d_acc_phi, *d_mass, *d_vel;
        float *h_pos = (float*) malloc(n_particles * 3 * sizeof(float));
        float *h_vel = (float*) malloc(n_particles * 3 * sizeof(float));
        float *h_acc_phi = (float*) malloc(n_particles * 4 * sizeof(float));

        hipMalloc(&d_h2pos,n_particles * 2 * sizeof(half2));
        hipMalloc(&d_hmass,n_particles * sizeof(half));
        hipMalloc(&d_acc_phi,n_particles * 4 * sizeof(float));
        hipMalloc(&d_pos,n_particles * 3 * sizeof(float));
        hipMalloc(&d_vel,n_particles * 3 * sizeof(float));
        hipMalloc(&d_mass,n_particles * sizeof(float));

        first = omp_get_wtime();
        hipMemcpy(d_pos,input_pos,n_particles * 3 * sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_vel,input_vel,n_particles * 3 * sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_mass,input_mass,n_particles * sizeof(float),hipMemcpyHostToDevice);

        copyFloat2Half2<<<numBlocks,blockSize>>>(d_pos,d_h2pos);
        copyFloat2Half_dim1<<<numBlocks,blockSize>>>(d_mass,d_hmass);
        second = omp_get_wtime();
        *copyTime += second-first;

        hipFree(d_mass);

        size_t shared_mem_size = blockSize * 2 * sizeof(half2);

        switch(solver){

            case 0:
                force_solve_cheap<<<numBlocks,blockSize>>>(d_h2pos,d_hmass,d_acc_phi,G,eps,n_particles);
                break;
            case 1:
                force_solve_cheap_shared_mem<<<numBlocks,blockSize,shared_mem_size>>>(d_h2pos,d_hmass,d_acc_phi,G,eps,n_particles);
                break;

        }

        hipDeviceSynchronize();

        first = omp_get_wtime();
        hipMemcpy(h_pos,d_pos,n_particles * 3 * sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_vel,d_vel,n_particles * 3 * sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_acc_phi,d_acc_phi,n_particles * 4 * sizeof(float),hipMemcpyDeviceToHost);
        second = omp_get_wtime();
        *copyTime += second-first;

        for (int step = 0; step < steps; step++){

            fast_add_4to3<<<numBlocks,blockSize>>>(d_acc_phi,d_vel,0.5 * dt);
            fast_add_3to3<<<numBlocks,blockSize>>>(d_vel,d_pos,1 * dt);

            copyFloat2Half2<<<numBlocks,blockSize>>>(d_pos,d_h2pos);

            switch(solver){

                case 0:
                    force_solve_cheap<<<numBlocks,blockSize>>>(d_h2pos,d_hmass,d_acc_phi,G,eps,n_particles);
                    break;
                case 1:
                    force_solve_cheap_shared_mem<<<numBlocks,blockSize,shared_mem_size>>>(d_h2pos,d_hmass,d_acc_phi,G,eps,n_particles);
                    break;

            }

            fast_add_4to3<<<numBlocks,blockSize>>>(d_acc_phi,d_vel,0.5 * dt);

            first = omp_get_wtime();
            save(h_pos,h_vel,h_acc_phi,n_particles,fp);
            second = omp_get_wtime();
            *saveTime += second-first;

            hipDeviceSynchronize();

            first = omp_get_wtime();
            hipMemcpy(h_pos,d_pos,n_particles * 3 * sizeof(float),hipMemcpyDeviceToHost);
            hipMemcpy(h_vel,d_vel,n_particles * 3 * sizeof(float),hipMemcpyDeviceToHost);
            hipMemcpy(h_acc_phi,d_acc_phi,n_particles * 4 * sizeof(float),hipMemcpyDeviceToHost);
            second = omp_get_wtime();
            *copyTime += second-first;

        }

        first = omp_get_wtime();
        save(h_pos,h_vel,h_acc_phi,n_particles,fp);
        second = omp_get_wtime();
        *saveTime += second-first;

        free(h_pos);
        free(h_acc_phi);

        hipFree(d_h2pos);
        hipFree(d_pos);
        hipFree(d_acc_phi);
        hipFree(d_hmass);

        total_second = omp_get_wtime();

        *totalTime = total_second - total_first;
        
    }
}