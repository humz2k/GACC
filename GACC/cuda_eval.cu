#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <omp.h>
#include "cuda_eval.h"
#include "kernels/kernels.h"

using namespace std;

void save(DATA_TYPE* pos, DATA_TYPE* vel, DATA_TYPE* phi_acc, int n_particles, std::ofstream &out){

    for (int i = 0; i < n_particles; i++){

        out.write( reinterpret_cast<char*>( &pos[i*3]), sizeof( DATA_TYPE ));
        out.write( reinterpret_cast<char*>( &pos[i*3 + 1]), sizeof( DATA_TYPE ));
        out.write( reinterpret_cast<char*>( &pos[i*3 + 2]), sizeof( DATA_TYPE ));

        out.write( reinterpret_cast<char*>( &vel[i*3]), sizeof( DATA_TYPE ));
        out.write( reinterpret_cast<char*>( &vel[i*3 + 1]), sizeof( DATA_TYPE ));
        out.write( reinterpret_cast<char*>( &vel[i*3 + 2]), sizeof( DATA_TYPE ));

        out.write( reinterpret_cast<char*>( &phi_acc[i*4]), sizeof( DATA_TYPE ));
        out.write( reinterpret_cast<char*>( &phi_acc[i*4 + 1]), sizeof( DATA_TYPE ));
        out.write( reinterpret_cast<char*>( &phi_acc[i*4 + 2]), sizeof( DATA_TYPE ));
        out.write( reinterpret_cast<char*>( &phi_acc[i*4 + 3]), sizeof( DATA_TYPE ));

    }

}

extern "C" { 
    void cuda_evaluate(DATA_TYPE* input_pos, DATA_TYPE* input_vel, DATA_TYPE* input_mass, int n_particles, int steps, DATA_TYPE G, DATA_TYPE eps, DATA_TYPE dt, int n_params, int solver, int v, double *saveTime, double *totalTime, double *copyTime){

        double first,second;
        double total_first,total_second;

        hipFree(0);

        *saveTime = 0;
        *totalTime = 0;
        *copyTime = 0;

        total_first = omp_get_wtime();

        std::ofstream out;
        out.open( "out.dat", std::ios::out | std::ios::binary);
        std::ofstream &fp = out;

        int blockSize = n_particles;
        if (blockSize > 256){
            
            if ((n_particles % 256) == 0){
                blockSize = 256;
            } else if ((n_particles % 128) == 0){
                blockSize = 128;
            } else if ((n_particles % 64) == 0){
                blockSize = 64;
            } else if ((n_particles % 32) == 0){
                blockSize = 32;
            }

        }
        int numBlocks = (n_particles + blockSize - 1) / blockSize;

        if (v){
            cout << "numBlocks" << numBlocks << endl;
            cout << "blockSize" << blockSize << endl;
        }

        DATA_TYPE *h_pos = (DATA_TYPE*) malloc(n_particles * 3 * sizeof(DATA_TYPE));
        DATA_TYPE *h_acc_phi = (DATA_TYPE*) malloc(n_particles * 4 * sizeof(DATA_TYPE));
        DATA_TYPE *h_vel = (DATA_TYPE*) malloc(n_particles * 3 * sizeof(DATA_TYPE));

        DATA_TYPE *d_pos;
        hipMalloc(&d_pos,n_particles * 3 * sizeof(DATA_TYPE));

        DATA_TYPE *d_acc_phi;
        hipMalloc(&d_acc_phi,n_particles * 4 * sizeof(DATA_TYPE));

        DATA_TYPE *d_vel;
        hipMalloc(&d_vel,n_particles * 3 * sizeof(DATA_TYPE));

        DATA_TYPE *d_mass;
        hipMalloc(&d_mass,n_particles * sizeof(DATA_TYPE));

        first = omp_get_wtime();
        hipMemcpy(d_pos,input_pos,n_particles * 3 * sizeof(DATA_TYPE),hipMemcpyHostToDevice);
        hipMemcpy(d_vel,input_vel,n_particles * 3 * sizeof(DATA_TYPE),hipMemcpyHostToDevice);
        hipMemcpy(d_mass,input_mass,n_particles * sizeof(DATA_TYPE),hipMemcpyHostToDevice);
        second = omp_get_wtime();
        *copyTime += second-first;

        size_t shared_mem_size = blockSize * 4 * sizeof(DATA_TYPE);

        switch (solver){

            case 0:
                force_solve_gpu<<<numBlocks,blockSize>>>(d_pos,d_mass,d_acc_phi,G,eps,n_particles);
                break;
            
            case 1:
                force_solve_shared_mem<<<numBlocks,blockSize,shared_mem_size>>>(d_pos,d_mass,d_acc_phi,G,eps,n_particles);
                break;

        }

        hipDeviceSynchronize();

        first = omp_get_wtime();
        hipMemcpy(h_pos,d_pos,n_particles * 3 * sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_vel,d_vel,n_particles * 3 * sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_acc_phi,d_acc_phi,n_particles * 4 * sizeof(DATA_TYPE),hipMemcpyDeviceToHost);
        second = omp_get_wtime();
        *copyTime += second-first;

        *saveTime += second-first;


        for (int step = 0; step < steps; step++){

            fast_add_4to3<<<numBlocks,blockSize>>>(d_acc_phi,d_vel,0.5 * dt);
            fast_add_3to3<<<numBlocks,blockSize>>>(d_vel,d_pos,1 * dt);

            switch (solver){

                case 0:
                    force_solve_gpu<<<numBlocks,blockSize>>>(d_pos,d_mass,d_acc_phi,G,eps,n_particles);
                    break;
                
                case 1:
                    force_solve_shared_mem<<<numBlocks,blockSize,shared_mem_size>>>(d_pos,d_mass,d_acc_phi,G,eps,n_particles);
                    break;

            }

            fast_add_4to3<<<numBlocks,blockSize>>>(d_acc_phi,d_vel,0.5 * dt);

            first = omp_get_wtime();
            save(h_pos,h_vel,h_acc_phi,n_particles,fp);
            second = omp_get_wtime();
            *saveTime += second-first;

            hipDeviceSynchronize();

            first = omp_get_wtime();
            hipMemcpy(h_pos,d_pos,n_particles * 3 * sizeof(DATA_TYPE),hipMemcpyDeviceToHost);
            hipMemcpy(h_vel,d_vel,n_particles * 3 * sizeof(DATA_TYPE),hipMemcpyDeviceToHost);
            hipMemcpy(h_acc_phi,d_acc_phi,n_particles * 4 * sizeof(DATA_TYPE),hipMemcpyDeviceToHost);
            second = omp_get_wtime();
            *copyTime += second-first;

        }

        first = omp_get_wtime();
        save(h_pos,h_vel,h_acc_phi,n_particles,fp);
        second = omp_get_wtime();
        *saveTime += second-first;

        hipFree(d_pos);
        hipFree(d_vel);
        hipFree(d_acc_phi);
        hipFree(d_mass);

        free(h_pos);
        free(h_vel);
        free(h_acc_phi);

        total_second = omp_get_wtime();

        *totalTime = total_second - total_first;
    }
}